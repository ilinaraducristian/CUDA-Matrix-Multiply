
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

using namespace std;

void print3DArray(float ***a, int n) {
  for(int i=0;i<n;i++) {
    for(int j=0;j<n;j++) {
      for(int k=0;k<n;k++) {
        cout<<a[i][j][k]<<' ';
      }
      cout<<endl;
    }
    cout<<endl<<endl<<endl;
  }
}

void print2DArray(float **a, int n) {
  for(int i=0;i<n;i++) {
    for(int j=0;j<n;j++) {
      cout<<a[i][j]<<' ';
    }
    cout<<endl;
  }
}

__global__ void multiply(float ***a, float ***b, int n) {

  int k = threadIdx.x+blockIdx.x;
  int j = threadIdx.y+blockIdx.y;
  int i = threadIdx.z+blockIdx.z;

  if(i<n && j<n && k<n) {
    a[i][j][k] = a[0][j][i] * b[0][i][k];
  }

}

__global__ void add(float ***a, int n) {

  int k = threadIdx.x+blockIdx.x;
  int j = threadIdx.y+blockIdx.y;
  int i = threadIdx.z+blockIdx.z;

  if(i<n && j<n && k<n) {
    for(int l=1;l<n;l++) {
      a[0][j][k]+=a[l][j][k];
    }
  }

}

int main(void) {
  int n = 100;
  // int blockSize = 256;
  // int blocks = (N + blockSize - 1) / blockSize;

  float ***a, ***b;
  hipMallocManaged(&a, n*sizeof(float**));
  hipMallocManaged(&b, n*sizeof(float**));

  for(int i=0;i<n;i++) {
    hipMallocManaged(&(a[i]), n*sizeof(float*));
    hipMallocManaged(&(b[i]), n*sizeof(float*));
  }

  for(int i=0;i<n;i++) {
      for(int j=0;j<n;j++) {
        hipMallocManaged(&(a[i][j]), n*sizeof(float));
        hipMallocManaged(&(b[i][j]), n*sizeof(float));
      }
  }

  for(int i=0;i<n;i++) {
    for(int j=0;j<n;j++) {
      a[0][i][j] = rand() % 4 + 1;
      b[0][i][j] = rand() % 4 + 1;
    }
  }

  // Print A, B

  cout<<"A:"<<endl;

  print2DArray(*a, n);

  cout<<"B:"<<endl;

  print2DArray(*b, n);

  // Multiply matrices
  multiply<<< 10, dim3(10, 10, 10) >>>(a, b, n);

  // Set number of threads and blocks

//   int blockSize = 1024;
//   int numBlocks = sqrt(n / blockSize);

//   addSquareMatrices<<< 1, dim3(32, 32, 32) >>>(x, y, n);

  hipDeviceSynchronize();

  add<<< 1, dim3(10, 10, 10) >>>(a, n);

  hipDeviceSynchronize();

  // Print A*B

  cout<<"A*B:"<<endl;

  print2DArray(*a, n);

  cout<<"done"<<endl;

  hipFree(a);
  hipFree(b);
  
  return 0;
}